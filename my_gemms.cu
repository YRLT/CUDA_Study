#include "hip/hip_runtime.h"
#include<stdio.h>

#include<vector>
#include<iostream>

#include<windows.h>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hipblas.h"
#include ""

#include "utils.cuh"

using namespace std;

float run_cublas_gemm(int dev, 
    const float A[][N],
    const float B[][N],
    float C[][N]) {
    hipblasHandle_t handle;

    float alpha, beta;

    alpha = 1.f;    beta = 0.f;

    // create cuBLAS handle
    hipblasCreate(&handle);
    hipError_t cudastat = hipSetDevice(dev);
    
    float* pCublasA = nullptr;
    float* pCublasB = nullptr;
    float* pCublasC = nullptr;

    cudastat = hipMalloc((void**)&pCublasA, N * N * sizeof(float));
    cudastat = hipMalloc((void**)&pCublasB, N * N * sizeof(float));
    cudastat = hipMalloc((void**)&pCublasC, N * N * sizeof(float));

    hipMemcpy(pCublasA, A, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(pCublasB, B, N * N * sizeof(float), hipMemcpyHostToDevice);

    float time_elapsed = 0;
    CudaStamp *cublas_stamp = set_clock_start();

    // Gemm
    hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T,
        N, N, N, &alpha, pCublasA, N, pCublasB, N, &beta, pCublasC, N);

    
    time_elapsed = set_clock_end(cublas_stamp, time_elapsed);
    
    hipDeviceSynchronize();
    
    delete cublas_stamp;

    hipMemcpy(C, pCublasC, N * N * sizeof(float), hipMemcpyDeviceToHost);
    

    hipblasDestroy(handle);
    hipFree(pCublasA);    hipFree(pCublasB);    hipFree(pCublasC);

    return time_elapsed;
 
}

__global__ void my_tiling_gemm(
    float* dev_a,
    float* dev_b,
    float* dev_c) {
    // dims: block( N / 16, N / 4), threads(32, 2)
    
    __shared__ float B_vecs[N][4]; // bandwidth: 128bit
    float thread_res[4] = {0,0,0,0};
    __shared__ float res[16][16]; // 16 row per block

    int col_begin = blockIdx.y * 4;
    int row_begin = blockIdx.x * 16;

    // read B tile into SM by block
    const int B_tile_nums = N / 16;
    const int row_in_Btile = threadIdx.y * 8 + threadIdx.x / 4;
    const int col_in_Btile = threadIdx.x % 4;

    for (int B_tile = 0; B_tile < B_tile_nums;  B_tile++) {
        //printf("%d", B_tile);
        B_vecs[B_tile * 16 + row_in_Btile][col_in_Btile] = dev_b[(B_tile * 16 + row_in_Btile) * N + col_begin + col_in_Btile];
    }
   
    __syncthreads();

    // mm with reading A_vec_tile, 4 threads per row
    const int A_tile_nums = N / 4;
    const int row_in_Atile = threadIdx.y * 8 + threadIdx.x / 4;
    const int col_in_Atile = threadIdx.x % 4;

    for (int A_tile = 0; A_tile < A_tile_nums; A_tile++) {
        thread_res[0] += dev_a[row_begin * N + (A_tile * 4 + col_in_Atile)] * B_vecs[(A_tile * 4 + col_in_Atile)][0];
        thread_res[1] += dev_a[row_begin * N + (A_tile * 4 + col_in_Atile)] * B_vecs[(A_tile * 4 + col_in_Atile)][1];
        thread_res[2] += dev_a[row_begin * N + (A_tile * 4 + col_in_Atile)] * B_vecs[(A_tile * 4 + col_in_Atile)][2];
        thread_res[3] += dev_a[row_begin * N + (A_tile * 4 + col_in_Atile)] * B_vecs[(A_tile * 4 + col_in_Atile)][3];
    }
    
    res[row_in_Atile][col_in_Atile * 4] = thread_res[0];
    res[row_in_Atile][col_in_Atile * 4 + 1] = thread_res[1];
    res[row_in_Atile][col_in_Atile * 4 + 2] = thread_res[2];
    res[row_in_Atile][col_in_Atile * 4 + 3] = thread_res[3];

    __syncthreads();
    // reduce result by single thread and save in first 4 cols in SM
    
    res[row_in_Atile][col_in_Atile * 4] += res[row_in_Atile][col_in_Atile * 4 + 1] + 
        res[row_in_Atile][col_in_Atile * 4 + 2] + res[row_in_Atile][col_in_Atile * 4 + 3];

    dev_c[(row_begin + row_in_Atile) + (col_begin + col_in_Atile) * N] = res[row_in_Atile][col_in_Atile * 4];
     
    return;
}


__device__ void sliced_mm_per_thread(float sliced_a[32][32],
    float sliced_b[32][32],
    float sliced_c[32][32],
    int thread_id) {
    // sliced_a and sliced_b both have been on shared mem
    // (i,j,k) -> {c[i,j] += a[i,k]*b[k,j],  i,k:0..64, }
    
    for (int i = 0; i < 32; i++) {
        sliced_c[i][thread_id] = 0;
        for (int k = 0; k < 32; k++) {
            sliced_c[i][thread_id] += sliced_a[i][k] * sliced_b[k][thread_id];
        }
    }

    return;
}

__global__ void my_balanced_tiling_gemm(
    float* dev_a,
    float* dev_b,
    float* dev_c) {
    // dims: block( N / 32, N / 32), threads(32)

    const int chunk_size = 32;
    const int C_chunk_x = blockIdx.x;
    const int C_chunk_y = blockIdx.y;

    __shared__ float A[chunk_size][chunk_size];
    __shared__ float B[chunk_size][chunk_size];
    __shared__ float C[chunk_size][chunk_size];

    int A_i = C_chunk_x;
    int B_j = C_chunk_y;
    
    int A_j = 0; int B_i = 0;

    for (int k = 0; k < gridDim.x; k++) {
        A_j = k;
        B_i = k;
        
        for (int row = 0; row < chunk_size; row++) {
            A[row][threadIdx.x] = dev_a[(A_i * chunk_size + row) * N + A_j * chunk_size + threadIdx.x];
            B[row][threadIdx.x] = dev_b[(B_i * chunk_size + row) * N + B_j * chunk_size + threadIdx.x];
        }

        sliced_mm_per_thread(A, B, C, threadIdx.x);

        for (int row = 0; row < chunk_size; row++) {
            dev_c[(C_chunk_y * chunk_size + threadIdx.x) * N + C_chunk_x * chunk_size + row] += C[row][threadIdx.x];
        }
    }
 
    return;
}

__global__ void my_simple_gemm(
    float* dev_a,
    float* dev_b,
    float* dev_c,
    int threads_size
){
    const float co_i = 1.0 / N;

    int threadid = threadIdx.x + blockDim.x * (blockIdx.x + blockIdx.y * gridDim.x);
    float res = 0.0;
    int i = 0, j = 0;

    int idx = threadid;

    for (; idx < N * N;) {
        i = floor(idx * co_i);
        j = idx - i * N;
        for (int k = 0; k < N; k++) {
             res += dev_a[i * N + k] * dev_b[k * N + j];
        }
        dev_c[j * N + i] = res;
        idx = idx + threads_size;
    }
    
    return ;
}

 float run_my_gemm(int dev,
     const float A[][N],
     const float B[][N],
     float C[][N]) {
     
     hipError_t cudastat = hipSetDevice(dev);

     float* pCudaA = nullptr;
     float* pCudaB = nullptr;
     float* pCudaC = nullptr;

     hipError_t cudaStatus;

     cudaStatus = hipMalloc((void**)&pCudaA, N * N * sizeof(float));
     cudaStatus = hipMalloc((void**)&pCudaB, N * N * sizeof(float));
     cudaStatus = hipMalloc((void**)&pCudaC, N * N * sizeof(float));
 
     cudaStatus = hipMemcpy(pCudaA, A, N * N * sizeof(float), hipMemcpyHostToDevice);

     if (pCudaA == NULL) {
         printf("couldn't allocate pCudaA GPU memory in my Gemm\n");
         return -1;
     }

     cudaStatus = hipMemcpy(pCudaB, B, N * N * sizeof(float), hipMemcpyHostToDevice);

     if (pCudaB == NULL) {
         printf("couldn't allocate pCudaB GPU memory in my Gemm\n");
         return  -1;
     }

     float time_elapsed = 0;
     CudaStamp* cublas_stamp = set_clock_start();

     dim3 grid(N/32, N/32);
     dim3 block(32);
 
     my_balanced_tiling_gemm << <grid, block >> > (pCudaA, pCudaB, pCudaC);

     //if (N * N < 1024 * 6 * 2) {
     //    int threads = floor(N * N /(2 * 6));
     //    dim3 grid(2, 6);
     //    dim3 block(threads);
     //    my_simple_gemm << <grid, block >> > (pCudaA, pCudaB, pCudaC, 2 * 6 * threads);
     //}
     //else{
     //    dim3 grid(2, 6);
     //    dim3 block(1024);
     //    my_simple_gemm << <grid, block >> > (pCudaA, pCudaB, pCudaC, 2 * 6 * 1024);
     //}

     time_elapsed = set_clock_end(cublas_stamp, time_elapsed);

     hipDeviceSynchronize();


     delete cublas_stamp;

     hipMemcpy(C, pCudaC, N * N * sizeof(float), hipMemcpyDeviceToHost);

     hipFree(pCudaA); hipFree(pCudaB); hipFree(pCudaC);
     return time_elapsed;
}


int main() {
	int dev = 0;
    int iter = 15;

    cout << "test N = " << N << ", iter = " << iter << endl;

 
    float (*A)[N] = new float[N][N];
    float (*B)[N] = new float[N][N];
    float (*C)[N] = new float[N][N];
    float (*my_C)[N] = new float[N][N];

    srand(2019);

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            A[i][j] = rand() / float(2019);
            B[i][j] = rand() / float(2019);
            C[i][j] = 0;
            my_C[i][j] = 0;
        }
    }

    float avg_cublas_time = 0;

    for (int it = 0; it < iter; it++) {
        avg_cublas_time += run_cublas_gemm(dev, A, B, C);
        hipDeviceSynchronize();
    }
    cout << avg_cublas_time / iter << " --- cublas time " << endl;

    Sleep(1000);

    float my_gemm_time = 0;
    
    for (int it = 0; it < iter; it++) {
        my_gemm_time += run_my_gemm(dev, A, B, my_C);;
        hipDeviceSynchronize();
    }
    cout << my_gemm_time / iter << " --- my gemm time " << endl;

    //std::cout << "C:\n" << endl;
    //printMatrix(C);
    //std::cout << "myC:\n" << endl;
    //printMatrix(my_C);

    cout << "check result: " << is_array2D_equil(C, my_C) << endl;

    delete[] A;
    delete[] B;
    delete[] C;

    cout << "freed A, B, C points" << endl;

    //get_device_info();

	return 0;
}